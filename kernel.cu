#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib> 


const int sizePoint = 700;
const int sizeIndividum = 1000;
const int MutationProbability = 10;
const float MutationDispersion = 5.0f;
const int Polynom = 3;
const float randMaxCount = 20.0f;
const int maxGeneration = 30;

__global__ void errorsKernel(float* points, float* individs, float* errors, int Polynom, int sizePoint)					// ��������� ������ �� GPU
{

	int id = threadIdx.x;
	float ans = 0;
	int x = 1;
	for (int i = 0; i < sizePoint; i++)
	{
		for (int j = 0; j < Polynom; j++)
		{
			for (int k = 0; k < j; k++)
			{
				x *= i;
			}
			x *= individs[id * Polynom + j];
			ans += x;
			x = 1;
		}

		ans = points[i] - ans;
		errors[id] += sqrt(ans * ans);
		ans = 0;
	}
}


void testErrorsKernel(float* points, float* individs, float* errors, int Polynom, int sizePoint, int sizeIndividum)		// ��������� ������ �� CPU
{
	for (int id = 0; id < sizeIndividum; id++)
	{
		float ans = 0.0f;
		errors[id] = 0.0f;
		int x = 0;
		for (int i = 0; i < sizePoint; i++)
		{
			for (int j = 0; j < Polynom; j++)
			{
				x = pow(i, j);
				x *= individs[id * Polynom + j];
				ans += x;
				x = 0;
			}

			ans = points[i] - ans;
			errors[id] += sqrt(ans * ans);
			ans = 0;
		}
	}
}

float RandomFloat(float a, float b) {
	float random = ((float)rand()) / (float)RAND_MAX;
	float diff = b - a;
	float r = random * diff;
	return a + r;
}

void cpu() {
	float* pointsH = new float[sizePoint]; 
	for (int i = 0; i < sizePoint; i++)							// ������� ��������� ����� �����
	{
		pointsH[i] = RandomFloat(0, randMaxCount);
	}

	float* individumsH = new float[sizeIndividum * Polynom];
	for (int i = 0; i < sizeIndividum * Polynom; i++)			// ������� ������ ���������
	{
		individumsH[i] = RandomFloat(0, randMaxCount);			
	}

	float* errorsH = new float[sizeIndividum];
	for (int i = 0; i < sizeIndividum; i++)
	{
		errorsH[i] = 1000;
	}

	unsigned int start_time_cpu = clock(); 

	for (int generation = 0; generation < maxGeneration; generation++)
	{
		testErrorsKernel(pointsH, individumsH, errorsH, Polynom, sizePoint, sizeIndividum);

		float* errorsCrossOver = new float[sizeIndividum];

		for (size_t i = 0; i != sizeIndividum; ++i)
			errorsCrossOver[i] = errorsH[i];
		sort(errorsCrossOver, errorsCrossOver + sizeIndividum);

		int merodianCrossOvering = sizeIndividum / 2;
		float merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		float* theBestInd = new float[Polynom];

		for (size_t i = 0; i < sizeIndividum; i++)			// ����� ���������
		{
			if (merodianErrorCrossOvering < errorsH[i]) {
				for (size_t j = 0; j < Polynom; j++)
				{
					individumsH[i * Polynom + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < Polynom; j++)
				{
					theBestInd[j] = individumsH[i * Polynom + j];
				}
			}
		}

		printf("Error = %f\n", errorsCrossOver[0]);

		for (int i = 0; i < sizeIndividum * Polynom; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % Polynom];
			}

			if (MutationProbability > (rand() % 100 + 1)) {
				individumsH[i] += RandomFloat(-MutationDispersion, MutationDispersion);
			}
		}
	}
	unsigned int end_time_cpu = clock(); 
	unsigned int search_time_cpu = end_time_cpu - start_time_cpu;
	printf("Time CPU = %i\n", search_time_cpu);
}

void gpu() {
	float* pointsH = new float[sizePoint];
	for (int i = 0; i < sizePoint; i++)				// ������� ��������� ����� �����
	{  
		pointsH[i] = RandomFloat(0, randMaxCount);
	}

	float* individumsH = new float[sizeIndividum * Polynom];
	for (int i = 0; i < sizeIndividum * Polynom; i++)	// ������� ������ ���������
	{
		individumsH[i] = RandomFloat(0, randMaxCount);
	}

	float* errorsH = new float[sizeIndividum];
	for (int i = 0; i < sizeIndividum; i++)
	{
		errorsH[i] = 1000;
	}

	unsigned int start_time_gpu = clock();
	float* pointsD = NULL;
	float* individumsD = NULL;
	float* errorsD = NULL;

	for (int generation = 0; generation < maxGeneration; generation++)
	{

		int sizeIndividumBytes = sizeIndividum * Polynom * sizeof(float);
		int sizePointBytes = sizePoint * sizeof(float);

		hipMalloc((void**)&pointsD, sizePointBytes);
		hipMalloc((void**)&individumsD, sizeIndividumBytes * Polynom);
		hipMalloc((void**)&errorsD, sizeIndividum * sizeof(float));

		hipMemcpy(pointsD, pointsH, sizePointBytes, hipMemcpyHostToDevice);
		hipMemcpy(individumsD, individumsH, sizeIndividumBytes, hipMemcpyHostToDevice);
		hipMemcpy(errorsD, errorsH, sizeIndividumBytes, hipMemcpyHostToDevice);

		errorsKernel << <1, sizeIndividum >> > (pointsD, individumsD, errorsD, Polynom, sizePoint);

		hipMemcpy(errorsH, errorsD, sizeIndividum * sizeof(float), hipMemcpyDeviceToHost);

		//----------------------
		float* errorsCrossOver = new float[sizeIndividum];

		for (size_t i = 0; i != sizeIndividum; ++i)
			errorsCrossOver[i] = errorsH[i];
		sort(errorsCrossOver, errorsCrossOver + sizeIndividum);
		printf("Error = %f\n", errorsCrossOver[0]);
		int merodianCrossOvering = sizeIndividum / 2;
		float merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		float* theBestInd = new float[Polynom];

		for (size_t i = 0; i < sizeIndividum; i++)				// ����� ���������
		{
			if (merodianErrorCrossOvering < errorsH[i]) {
				for (size_t j = 0; j < Polynom; j++)
				{
					individumsH[i * Polynom + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < Polynom; j++)
				{
					theBestInd[j] = individumsH[i * Polynom + j];
				}
			}
		}

		for (int i = 0; i < sizeIndividum * Polynom; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % Polynom];
			}

			if (MutationProbability > (rand() % 100 + 1)) {
				individumsH[i] += RandomFloat(-MutationDispersion, MutationDispersion);
			}
		}
	}
	unsigned int end_time_gpu = clock();
	unsigned int search_time_gpu = end_time_gpu - start_time_gpu;

	printf("Time GPU = %i\n", search_time_gpu);
	
	hipFree(pointsD);
	hipFree(individumsD);
	hipFree(errorsD);

	delete pointsH;
	delete individumsH;
	delete errorsH;
}

int main()
{
	cpu();
	gpu();
	system("pause");
	return 0;
}
